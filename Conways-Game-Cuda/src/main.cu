#include "hip/hip_runtime.h"
//
// Created by pascal on 19.01.2020
//

#define N 4
#define M 4

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

__global__ void iterate(float *source, float *target, int width, int height)
{
    int x = blockId.x * blockDim.x + threadID.x;
    int y = blockId.y * blockDim.y + threadID.y;

    if (x < width && y < height) {
        int neighbours = 0;

        for (int dx = -1; dx <= 1; ++dx) {
            for (int dy = -1; dy <= 1; ++dy) {
                int xdx = x + dx, ydy = y + dy;

                if (dx != 0 || dy != 0) {
                    if (0 <= xdx && xdx < width && 0 <= ydy && ydy < height) {
                        int idx = getIndex(xdx, ydy, width, height);
                        neighbours += (source[idx] > 0);
                    }
                }
            }
        }

        int idx = getIndex(x, y, width, height);
        target[idx] = (neighbours == 3 || (source[idx] > 0 && neighbours == 2));
    }
}

__device__ int getIndex(int x, int y, int width, int height) {
    return y * width + x;
}

int main(int argc, char const *argv[])
{
    thrust::host_vector<int> bufferA_host(N*M, 0), bufferB_host(N*M, 0);
    thrust::device_vector<int> bufferA_device = bufferA_host, bufferB_device = bufferB_host;


    

    /* code */
    return 0;
}
